
#include <hip/hip_runtime.h>
// #include <util/cuda_managed.h>
//
// #include <cuda_runtime.h>
//
// namespace util {
//     __host__ void* CudaManaged::operator new(size_t len)
//     {
//         void* ptr;
//         cudaMallocManaged(&ptr, len);
//         cudaDeviceSynchronize();
//         return ptr;
//     }
//
//     /**
//      * @brief rewrited delete operator for cuda managed object
//      * @note cudaDeviceSynchronize in device code is deprecated so using new/delete in
//      * device code always causes problem now. But we can still use stack object safely.
//      */
//     __host__ __device__ void CudaManaged::operator delete(void* ptr)
//     {
//     #ifndef __CUDA_ARCH__
//         cudaDeviceSynchronize();
//     #endif
//         cudaFree(ptr);
//     }
// }
